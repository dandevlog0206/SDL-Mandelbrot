#include "hip/hip_runtime.h"
#include "mandelbrot_cuda.h"

#include <hip/hip_runtime.h>
#include <>
#include "color.h"

using namespace std;

using real_t    = Mandelbrot::real_t;
using PixelInfo = Mandelbrot::PixelInfo;

template <class T>
__device__ inline uint32_t mandelbrot(T& cx, T& cy, uint32_t max_iter);
__device__ inline uint32_t make_color(uint8_t r, uint8_t g, uint8_t b, uint8_t a);
__device__ inline uint32_t lerp_color(uint32_t a, uint32_t b, double t);

__device__ inline void acc_color(uint32_t col, uint32_t& r, uint32_t& g, uint32_t& b) {
	struct Color { uint8_t b, g, r, a; } color = *(Color*)&col;

	r += color.r;
	g += color.g;
	b += color.b;
}

static __device__ __forceinline__ uint32_t lcg(uint32_t& prev)
{
	prev = (1664525u * prev + 1013904223u);
	return prev & 0x00FFFFFF;
}

static __device__ __forceinline__ float rnd(uint32_t& prev)
{
	return ((float)lcg(prev) / (float)0x01000000);
}

struct Constants {
	int      width;
	int      height;
	uint32_t iter;

	uint32_t color_idx;
	real_t   color_scale;
	bool     color_smooth;

	bool     stop_all;
};

extern "C" {
	__constant__ uint32_t cuda_colormap[6][256];
	__constant__  Constants params;
}

template <class T>
__device__ uint32_t get_color_idx(T iter) {
	return (uint32_t)(params.color_scale * 256 * iter / params.iter) % 256;
}

__global__ void mandelbrot_kernel(
	uint32_t* device_surface,
	PixelInfo* device_pixel_info,
	uint32_t   sample_total,
	uint32_t   sample,
	real_t     min_x,
	real_t     max_y,
	real_t     dp
) {
	if (params.stop_all) return;

	int w = blockDim.x * blockIdx.x + threadIdx.x;
	int h = blockDim.x * blockIdx.y + threadIdx.y;

	if (params.width <= w || params.height <= h) return;

	auto pixel_off  = h * params.width + w;
	auto& info      = *(device_pixel_info + pixel_off);

	if (info.sample_count >= sample_total) return;
 
	auto seed  = (uint32_t)clock();
	auto count = min(sample_total - info.sample_count, sample);

	uint32_t col;
	uint32_t r = info.acc_r;
	uint32_t g = info.acc_g;
	uint32_t b = info.acc_b;

	for (uint32_t i = 0; i < count; ++i) {
		if (params.stop_all) return;
		
		real_t cx = min_x + dp * (w + rnd(seed));
		real_t cy = max_y - dp * (h + rnd(seed));

		auto iterated = mandelbrot<real_t>(cx, cy, params.iter);

		if (iterated == params.iter) {
			col = 0xff000000;
		} else if (!params.color_smooth) {
			col = cuda_colormap[params.color_idx][get_color_idx(iterated)];
		} else {
			double log_zn  = log(cx * cx + cy * cy) / 2.;
			double nu      = log(log_zn / log(2.)) / log(2.);
			auto real_iter = iterated + 3.5 - nu;

			auto col1 = cuda_colormap[params.color_idx][get_color_idx(real_iter)];
			auto col2 = cuda_colormap[params.color_idx][get_color_idx(real_iter + 1.)];

			col = lerp_color(col1, col2, fmod(real_iter, 1.));
		}

		acc_color(col, r, g, b);
	}

	info.sample_count += count;
	info.acc_r         = r;
	info.acc_g         = g;
	info.acc_b         = b;

	auto& pixel = *(device_surface + pixel_off);
	pixel = make_color(r / info.sample_count, g / info.sample_count, b / info.sample_count);
}

MandelbrotCUDA::MandelbrotCUDA(SDL_Renderer* renderer) 
	: Mandelbrot(renderer)
{
	size_t size = surface->w * surface->h;

	hipHostRegister(surface->pixels, size * sizeof(uint32_t), hipHostRegisterDefault);
	hipHostRegister(render_info.pixels, size * sizeof(PixelInfo), hipHostRegisterDefault);

	hipMalloc((void**)&device_surface, size * sizeof(uint32_t));
	hipMalloc((void**)&device_pixel_info, size * sizeof(PixelInfo));

	hipMemcpyToSymbol(HIP_SYMBOL(cuda_colormap), colormap, sizeof(colormap));

	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);

	block_size        = 8;
	sample_total      = 1;
	sample_per_launch = 1;
	sample_count      = 0;

	update();
}

MandelbrotCUDA::~MandelbrotCUDA()
{
	hipHostUnregister(surface->pixels);
	hipHostUnregister(render_info.pixels);
	hipFree(device_surface);
	hipFree(device_pixel_info);
	hipStreamDestroy(streams[0]);
	hipStreamDestroy(streams[1]);
}

void MandelbrotCUDA::setTotalSample(uint32_t sample)
{
	auto clear = sample_count > (sample_total = sample);
	update(clear, clear);
}

void MandelbrotCUDA::draw()
{
	size_t size = surface->w * surface->h * sizeof(uint32_t);

	hipMemcpyAsync(surface->pixels, device_surface, size, hipMemcpyDeviceToHost, streams[0]);
	hipStreamSynchronize(streams[0]);
	Mandelbrot::draw();
}

void MandelbrotCUDA::stop()
{
	if (is_rendering) {
		size_t offset = offsetof(Constants, stop_all);

		hipMemcpyToSymbolAsync(HIP_SYMBOL(params), &is_rendering, 1, offset, hipMemcpyHostToDevice, streams[0]);
		wait();
		hipMemcpyToSymbolAsync(HIP_SYMBOL(params), &is_rendering, 1, offset, hipMemcpyHostToDevice, streams[0]);
	}
}

void MandelbrotCUDA::wait()
{
	hipStreamSynchronize(streams[1]);
}

void MandelbrotCUDA::resize()
{
	int width, height;
	SDL_GetWindowSize(window, &width, &height);
	
	size_t size = width * height;

	stop();
	hipFree(device_surface);
	hipFree(device_pixel_info);

	hipMalloc((void**)&device_surface, size * sizeof(uint32_t));
	hipMalloc((void**)&device_pixel_info, size * sizeof(PixelInfo));
	
	hipHostUnregister(surface->pixels);
	hipHostUnregister(render_info.pixels);

	Mandelbrot::resize();

	hipHostRegister(surface->pixels, size * sizeof(uint32_t), hipHostRegisterDefault);
	hipHostRegister(render_info.pixels, size * sizeof(PixelInfo), hipHostRegisterDefault);
}

void MandelbrotCUDA::move(int32_t rel_px, int32_t rel_py)
{
	stop();

	size_t size = surface->w * surface->h;

	hipMemcpy(surface->pixels, device_surface, size * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(render_info.pixels, device_pixel_info, size * sizeof(PixelInfo), hipMemcpyDeviceToHost);

	Mandelbrot::move(rel_px, rel_py);
	
	hipMemcpy(device_surface, surface->pixels, size * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(device_pixel_info, render_info.pixels, size * sizeof(PixelInfo), hipMemcpyHostToDevice);
}

void MandelbrotCUDA::drawSurface()
{
	dim3 grid((width - 1) / block_size + 1, (height - 1) / block_size + 1);
	dim3 block(block_size, block_size);

	real_t min_x = pos_x - 2. * scale * aspect;
	real_t max_y = pos_y + 2. * scale;
	real_t dp    = 4. * scale / height;

	while (sample_count < sample_total) {
		if (stop_all) break;

		mandelbrot_kernel<<<grid, block, 0, streams[1]>>> (
			device_surface,
			device_pixel_info,
			sample_total,
			sample_per_launch,
			min_x, max_y, dp);

		sample_count = min(sample_count + sample_per_launch, sample_total);
		hipStreamSynchronize(streams[1]);
	}

	hipStreamSynchronize(streams[1]);
}

void MandelbrotCUDA::update(bool rerender_all, bool clear_surface)
{
	Mandelbrot::update(rerender_all, clear_surface);
	
	sample_count = 0;
	
	size_t size = surface->w * surface->h;

	if (rerender_all)
		hipMemset(device_pixel_info, 0, size * sizeof(PixelInfo));
	if (clear_surface)
		hipMemset(device_surface, 0, size * sizeof(uint32_t));

	Constants constants = {
		width, height, iter, color_idx,color_scale, smooth, stop_all
	};

	hipMemcpyToSymbol(HIP_SYMBOL(params), &constants, sizeof(Constants));
}